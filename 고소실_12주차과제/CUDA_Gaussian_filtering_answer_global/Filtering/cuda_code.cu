#include "hip/hip_runtime.h"
#pragma once

#include "cuda_code.cuh"

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#define Window 2
#define BLOCK_SIZE (1 <<5)

__constant__ float constant_gaussian_kernel[ 25 ];

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ������� �ʴ´�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Gaussian_kernel_no_shared(IN unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	/*Todo*/

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int c = 0, k = 0, l = 0;
	double mean = 0.0;
	int w = floor((double)5 / 2);

	for (c = 0; c < width; c++) {
		mean = 0.0;
		for (k = -w; k <= w; k++) {
			for (l = -w; l <= w; l++) {
				if (id + k < 0 || c + l < 0 || id + k >= height || c + l >= width) mean += 0;
				else mean += d_bitmaps[(id + k) * width + c + l] * constant_gaussian_kernel[(w + k) * 5 + w + l];
			}
		}
		d_Gaussian[id * width + c] = mean;
	}


}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ unsigned char sharedBuffer[];
__global__ void Gaussian_kernel_shared(INOUT unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	/*Todo*/
	const unsigned block_id = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;

	int c = 0, k = 0, l = 0, i = 0;
	double mean = 0.0;
	int w = floor((double)5 / 2);
	int w0 = 5;

	if (thread_id == 0) { // thread_id on the upper part
		for (c = 0; c < width; c++) {
			for (k = 1; k < w + 1; k++) {
				if (id - k >= 0 && id - k <= height) sharedBuffer[(-k + w) * width + c] = d_bitmaps[(id - k) * width + c];
				else sharedBuffer[(-k + w) * width + c] = 0; // to reduce condition management
			}
		}
	}

	if (thread_id == BLOCK_SIZE - 1) { // thread_id on the below part
		for (c = 0; c < width; c++) {
			for (k = BLOCK_SIZE; k < BLOCK_SIZE + w; k++) {
				if (id + k - BLOCK_SIZE + 1 < height) sharedBuffer[(k + w) * width + c] = d_bitmaps[(id + k - BLOCK_SIZE + 1) * width + c];
				else sharedBuffer[(k + w) * width + c] = 0; // to reduce condition management
			}
		}
	}

	for (c = 0; c < width; c++) {
		sharedBuffer[(thread_id + w) * width + c] = d_bitmaps[(id) * width + c];
		// fill the line of the sharedBuffer (thread_id) with original data (id)
	}
	
	__syncthreads();
	// syncrhonization of threas

	for (c = 0; c < width; c++) {
		mean = 0.0;
		for (k = -w; k <= w; k++) {
			for (l = -w; l <= w; l++) {
				if (c + l < 0 || c + l >= width) mean += 0; // height conditions are already processed at the above part
				else mean += sharedBuffer[(thread_id + w + k) * width + c + l] * constant_gaussian_kernel[(w + k) * w0 + w + l];
				// gaussian filtering on the thread id + w
			}
		}
		d_Gaussian[(id) * width + c] = mean;
	}

}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Constant variable �� gaussian kernel�� �����ϴ� �Լ�
//	�Ŀ� gaussian filtering ���� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Set_Gaussain_Kernel(){
	float _1 = 1.0f / 256.0f;
	float _4 = _1 * 4;
	float _6 = _1 * 6;
	float _16 = _1 * 16;
	float _24 = _1 * 24;
	float _36 = _1 * 36;

	float *p_gaussian_kernel = new float[25];

	p_gaussian_kernel[0] = p_gaussian_kernel[4] = p_gaussian_kernel[20] = p_gaussian_kernel[24] = _1;
	p_gaussian_kernel[1] = p_gaussian_kernel[3] = p_gaussian_kernel[5] = p_gaussian_kernel[9]= _4;
	p_gaussian_kernel[15] = p_gaussian_kernel[19] = p_gaussian_kernel[21] = p_gaussian_kernel[23] = _4;
	p_gaussian_kernel[2] = p_gaussian_kernel[10] = p_gaussian_kernel[14] = p_gaussian_kernel[22] = _6;
	p_gaussian_kernel[6] = p_gaussian_kernel[8] = p_gaussian_kernel[16] = p_gaussian_kernel[18] = _16;
	p_gaussian_kernel[7] = p_gaussian_kernel[11] =p_gaussian_kernel[13] = p_gaussian_kernel[17] = _24;
	p_gaussian_kernel[12] = _36;

	hipMemcpyToSymbol(HIP_SYMBOL( constant_gaussian_kernel), p_gaussian_kernel, sizeof( float ) * 25 );

	delete[] p_gaussian_kernel;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	Shared_flag �Է� �� NO_SHARED �� SHARED �� �� ���� ��ũ�θ� ������
//	flag���� �´� Ŀ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

float Do_Gaussian_on_GPU(IN unsigned char *p_bitmaps, OUT unsigned char *p_Gaussian, long width, long height, int Shared_flag)
{
	/*Todo*/
	hipError_t cudaStatus;
	CUDA_CALL(hipSetDevice(0));

	unsigned char* d_bitmaps, * d_Gaussian;

	size_t mem_size;

	mem_size = width * height * sizeof(unsigned char);
	CUDA_CALL(hipMalloc(&d_bitmaps, mem_size));
	CUDA_CALL(hipMalloc(&d_Gaussian, mem_size));

	CUDA_CALL(hipMemcpy(d_bitmaps, p_bitmaps, mem_size, hipMemcpyHostToDevice));

	dim3 dimBlock(BLOCK_SIZE, 1);
	dim3 dimGrid(height / dimBlock.x, 1);

	Set_Gaussain_Kernel();
	int w = floor((double)5 / 2);
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();

	switch (Shared_flag)
	{
	case NO_SHARED:
		Gaussian_kernel_no_shared << < dimGrid, dimBlock >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	case SHARED:
		Gaussian_kernel_shared << < dimGrid, dimBlock, sizeof(unsigned char) * ((BLOCK_SIZE + 2 * w) * width) >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	}

	CUDA_CALL(cudaStatus = hipDeviceSynchronize());
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipMemcpy(p_Gaussian, d_Gaussian, mem_size, hipMemcpyDeviceToHost));
	hipFree(d_bitmaps);
	hipFree(d_Gaussian);

	return device_time;
}