#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;
#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#define	IN
#define OUT
#define INOUT

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define N_SIZE (1 << 26)													// ��ü ������ ������
#define NF_SIZE (1 << 6)													// Nf ũ��

#define BLOCK_SIZE (1 << 6)													// CUDA Ŀ�� thread block ������


#define BLOCK_WIDTH (1 << 3)
#define BLOCK_HEIGHT (BLOCK_SIZE / BLOCK_WIDTH)

#define N_ITERATION (1 << 0)												// ���� �ݺ� Ƚ��

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
int Nf;

int *h_ArrayElements;
int *h_SumOfArrayElements_CPU;
int *h_SumOfArrayElements_GPU;

hipError_t Sum_n_elements_GPU(IN int *p_ArrayElements, OUT int *p_SumOfElements_GPU, int Nf);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	�迭�� index - Nf ���� index + Nf ������ ������ ���� ����ϴ� Ŀ�� �ڵ�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Sum_n_elements_Kernel(IN int *d_ArrayElements, OUT int *d_SumOfArrayElements, int N, int Nf) {
    /*Todo*/

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    d_SumOfArrayElements[tid] = 0;
    for (int k = tid - Nf; k <= tid + Nf; k++) {
        if (k < 0 || k >= N) continue;
        d_SumOfArrayElements[tid] += d_ArrayElements[k];
    }

}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	�迭�� index - Nf ���� index + Nf ������ ������ ���� ����ϴ� C �ڵ�
//	GPU kernel�� ����� �񱳸� ���� ���� ����� �Ͽ����� �Ǵ��ϴ� �����ͷ� Ȱ��
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Sum_n_elements_CPU(IN int *p_ArrayElements, OUT int *p_SumOfElements_CPU, int Nf) {
    /*Todo*/
 
    for (int i = 0; i < N; i++) {
        p_SumOfElements_CPU[i] = 0;
        for (int k = i - Nf; k <= i + Nf; k++) {
            if (k < 0 || k >= N) continue;
            p_SumOfElements_CPU[i] += p_ArrayElements[k];
        }
    }
   
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	�־��� bin ������ �д� �ڵ�
//	ù 4����Ʈ�� ��ü �������� ����, ���� 4����Ʈ�� Nf�� ũ��, �� ���� N���� int�� �����Ͱ� ����
//	�����ʹ� -100 ~ 100 ������ ���� ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void read_bin_file() {
    printf("***Binary File Read Start!!\n");
    FILE *fp = fopen("Cuda_HW3_input.bin", "rb");
    fread(&N, sizeof(int), 1, fp);
    fread(&Nf, sizeof(int), 1, fp);

    h_ArrayElements = (int *)malloc(N * sizeof(int));
    h_SumOfArrayElements_CPU = (int *)malloc(N * sizeof(int));
    h_SumOfArrayElements_GPU = (int *)malloc(N * sizeof(int));

    fread(h_ArrayElements, sizeof(int), N, fp);

    fclose(fp);
    printf("***Binary File Read End!!\n\n");
}

void init_bin_file(IN int n, IN int nf) {
    printf("***Binary File Create Start!!\n");
    srand((unsigned)time(NULL));
    FILE *fp = fopen("Cuda_HW3_input.bin", "wb");
    fwrite(&n, sizeof(int), 1, fp);
    fwrite(&nf, sizeof(int), 1, fp);

    int i, input;

    for (i = 0; i < n; i++) {
        input = (int)((float)rand() / RAND_MAX * 200 - 100);
        fwrite(&input, sizeof(int), 1, fp);
    }

    fclose(fp);
    printf("***Binary File Create End!!\n\n");
}

int main()
{
    int i;
    init_bin_file(N_SIZE, NF_SIZE);
    read_bin_file();

    TIMER_T CPU_time = 0.0f, GPU_time_NO_SHARED = 0.0f;

    printf("N: %d, NF: %d, BLOCK_SIZE: %d\n", N, Nf, BLOCK_SIZE);

    for (i = 0; i < N_ITERATION; i++) {
        CHECK_TIME_START;
        Sum_n_elements_CPU(h_ArrayElements, h_SumOfArrayElements_CPU, Nf);
        CHECK_TIME_END(compute_time);
        CPU_time += compute_time;
     
        Sum_n_elements_GPU(h_ArrayElements, h_SumOfArrayElements_GPU, Nf);
     
        GPU_time_NO_SHARED += device_time;
    }

    for (i = 0; i < N; i++) {
        if (h_SumOfArrayElements_CPU[i] != h_SumOfArrayElements_GPU[i]) {
            printf("%d : CPU : %d,\tGPU : %d\n", i, h_SumOfArrayElements_CPU[i], h_SumOfArrayElements_GPU[i]);
            break;
        }
    }
    if (i == N)
        printf("***Kernel execution Success!!\n\n");

    printf("***CPU compute time : %.3f ms\n", CPU_time / N_ITERATION);
    printf("***GPU NO SHARED compute time : %.3f ms\n", GPU_time_NO_SHARED / N_ITERATION);


    // Write the output array into the output file.
    FILE *fp = fopen("Cuda_HW3_output.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N, sizeof(int), 1, fp);
    fwrite(&Nf, sizeof(int), 1, fp);

    for (i = 0; i < N; i++)
        fwrite(&h_SumOfArrayElements_GPU[i], sizeof(int), 1, fp);

    fclose(fp);



    free(h_ArrayElements);
    free(h_SumOfArrayElements_CPU);
    free(h_SumOfArrayElements_GPU);

    return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

hipError_t Sum_n_elements_GPU(IN int *p_ArrayElements, OUT int *p_SumOfElements_GPU, int Nf) {
    hipError_t cudaStatus;
    /*Todo*/
    CHECK_TIME_INIT_GPU();

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }/////////////  if(cu.....  ==CUDA_CALL
    /*Todo*/

    int(*d_Array), (*d_Sum);

    size_t size;

    size = N * sizeof(int);

    CUDA_CALL(hipMalloc(&d_Array, size))

        CUDA_CALL(hipMemcpy(d_Array, p_ArrayElements, size, hipMemcpyHostToDevice))


        CUDA_CALL(hipMalloc(&d_Sum, size))

    size = N;

    CHECK_TIME_START_GPU()
        // Assume that width and height are multiples of BLOCK SIZE.
        dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(N / dimBlock.x, 1);

    Sum_n_elements_Kernel << < dimGrid, dimBlock >> > (d_Array, d_Sum, N, Nf);

    CHECK_TIME_END_GPU(device_time)
    
    size = N * sizeof(int);

    CUDA_CALL(hipGetLastError())

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        CUDA_CALL(hipDeviceSynchronize())
        CUDA_CALL(hipMemcpy(p_SumOfElements_GPU, d_Sum, size, hipMemcpyDeviceToHost))


        CHECK_TIME_DEST_GPU();

Error:
    hipFree(d_Array);
    hipFree(d_Sum);

    return cudaStatus;
}
