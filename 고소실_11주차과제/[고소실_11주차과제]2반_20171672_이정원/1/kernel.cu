#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define BLOCK_SIZE 32
#define DATA_SIZE (1 << 20)

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N_EQUATIONS;
float* A, * B, * C;
float* X0, * X1;
float* FX0, * FX1;
float* X0_GPU, * X1_GPU;
float* FX0_GPU, * FX1_GPU;


hipError_t find_roots_GPU(float* A, float* B, float* C, float* X0_GPU, float* X1_GPU, float* FX0_GPU, float* FX1_GPU);

__global__ void find_roots_Kernel(float* d_A, float* d_B, float* d_C, float* d_X0, float* d_X1, float* d_FX0, float* d_FX1)
{

    int tid;

    tid = blockDim.x * blockIdx.x + threadIdx.x;
    float a, b, c, d, x0, x1, tmp;
    a = d_A[tid]; b = d_B[tid]; c = d_C[tid];
    d = sqrtf(b * b - 4.0f * a * c);
    tmp = 1.0f / (2.0f * a);
    d_X0[tid] = x0 = (-b - d) * tmp;
    d_X1[tid] = x1 = (-b + d) * tmp;
    d_FX0[tid] = (a * x0 + b) * x0 + c;
    d_FX1[tid] = (a * x1 + b) * x1 + c;

}


void find_roots_CPU(float* A, float* B, float* C, float* X0, float* X1, float* FX0, float* FX1, int n) {
    int i;
    float a, b, c, d, x0, x1, tmp;
    for (i = 0; i < n; i++) {
        a = A[i]; b = B[i]; c = C[i];
        d = sqrtf(b * b - 4.0f * a * c);
        tmp = 1.0f / (2.0f * a);
        X0[i] = x0 = (-b - d) * tmp;
        X1[i] = x1 = (-b + d) * tmp;
        FX0[i] = (a * x0 + b) * x0 + c;
        FX1[i] = (a * x1 + b) * x1 + c;
    }
}

void init_bin_file(int size) {
    srand((unsigned)time(NULL));
    printf("***Binary File init Start!!\n");
    FILE* fp = fopen("A.bin", "wb");

    fwrite(&size, sizeof(int), 1, fp);

    int random_num;
    for (int i = 0; i < size; i++) {
        random_num = (float)((float)rand() / RAND_MAX * 20.0f - 10.0f);
        fwrite(&random_num, sizeof(float), 1, fp);
    }

    fclose(fp);

    fp = fopen("B.bin", "wb");

    fwrite(&size, sizeof(int), 1, fp);

    for (int i = 0; i < size; i++) {
        random_num = (float)((float)rand() / RAND_MAX * 1000.0f + 100.0f);
        fwrite(&random_num, sizeof(float), 1, fp);
    }

    fclose(fp);

    fp = fopen("C.bin", "wb");

    fwrite(&size, sizeof(int), 1, fp);

    for (int i = 0; i < size; i++) {
        random_num = (float)((float)rand() / RAND_MAX * 20.0f - 10.0f);
        fwrite(&random_num, sizeof(float), 1, fp);
    }

    fclose(fp);


    printf("***Binary File init End!!\n\n");
}

void read_bin_file()
{
    printf("***Binary File Read Start!!\n");
    FILE* fp = fopen("A.bin", "rb");
    /*Todo*/

    fread(&N_EQUATIONS, sizeof(int), 1, fp);

    A = new float[N_EQUATIONS];
    B = new float[N_EQUATIONS];
    C = new float[N_EQUATIONS];
    X0 = new float[N_EQUATIONS];
    X1 = new float[N_EQUATIONS];
    FX0 = new float[N_EQUATIONS];
    FX1 = new float[N_EQUATIONS];
    X0_GPU = new float[N_EQUATIONS];
    X1_GPU = new float[N_EQUATIONS];
    FX0_GPU = new float[N_EQUATIONS];
    FX1_GPU = new float[N_EQUATIONS];


    fread(A, sizeof(float), N_EQUATIONS, fp);

    fclose(fp);

    fp = fopen("B.bin", "rb");
    fread(&N_EQUATIONS, sizeof(int), 1, fp);
    fread(B, sizeof(float), 1, fp);
    fclose(fp);

    fp = fopen("C.bin", "rb");
    fread(&N_EQUATIONS, sizeof(int), 1, fp);
    fread(C, sizeof(float), 1, fp);
    fclose(fp);

    printf("***Binary File Read End!!\n\n");
}

int main()
{
    init_bin_file(DATA_SIZE);
    read_bin_file();

    printf("N_EQUATIONS is %d.\n", N_EQUATIONS);
    int i;
    FILE* fp;

    //CPU 
    printf("***find_roots_CPU Start!!\n");
    CHECK_TIME_START;
    find_roots_CPU(A, B, C, X0, X1, FX0, FX1, N_EQUATIONS);
    CHECK_TIME_END(compute_time);
    printf("***find_roots_CPU End!!\n\n");

    printf("CPU time = %.6f\n", compute_time);
    //GPU

    printf("***find_roots_GPU Start!!\n");
    find_roots_GPU(A, B, C, X0_GPU, X1_GPU, FX0_GPU, FX1_GPU);
    printf("***find_roots_GPU End!!\n\n");

    printf("GPU time = %.6f\n", device_time);

    // Write the output array into the output file.
    
    fp = fopen("X0.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N_EQUATIONS, sizeof(int), 1, fp);

    for (i = 0; i < N_EQUATIONS; i++)
        fwrite(&X0_GPU[i], sizeof(float), 1, fp);

    fclose(fp);


    fp = fopen("X1.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N_EQUATIONS, sizeof(int), 1, fp);

    for (i = 0; i < N_EQUATIONS; i++)
        fwrite(&X1_GPU[i], sizeof(float), 1, fp);

    fclose(fp);


    fp = fopen("FX0.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N_EQUATIONS, sizeof(int), 1, fp);

    for (i = 0; i < N_EQUATIONS; i++)
        fwrite(&FX0_GPU[i], sizeof(float), 1, fp);

    fclose(fp);

    printf("end!!\n\n");


    fp = fopen("FX1.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N_EQUATIONS, sizeof(int), 1, fp);

    for (i = 0; i < N_EQUATIONS; i++)
        fwrite(&X0_GPU[i], sizeof(float), 1, fp);

    fclose(fp);
   

    return 0;
}


hipError_t find_roots_GPU(float* A, float* B, float* C, float* X0_GPU, float* X1_GPU, float* FX0_GPU, float* FX1_GPU)
{
    CHECK_TIME_INIT_GPU();

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }/////////////  if(cu.....  ==CUDA_CALL
    /*Todo*/

    float* d_A, * d_B, * d_C;
    float* d_X0, * d_X1;
    float* d_FX0, * d_FX1;

    size_t size;

    size = N_EQUATIONS * sizeof(float);

    CUDA_CALL(hipMalloc(&d_A, size))

        CUDA_CALL(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice))

    CUDA_CALL(hipMalloc(&d_B, size))

        CUDA_CALL(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice))

    CUDA_CALL(hipMalloc(&d_C, size))

        CUDA_CALL(hipMemcpy(d_C, C, size, hipMemcpyHostToDevice))


        CUDA_CALL(hipMalloc(&d_X0, size))
        CUDA_CALL(hipMalloc(&d_X1, size))
        CUDA_CALL(hipMalloc(&d_FX0, size))
        CUDA_CALL(hipMalloc(&d_FX1, size))

    
        size = N_EQUATIONS;

    CHECK_TIME_START_GPU()
    
        dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(N_EQUATIONS / dimBlock.x, 1);

    find_roots_Kernel << < dimGrid, dimBlock >> > (d_A, d_B, d_C, d_X0, d_X1, d_FX0, d_FX1);

    CHECK_TIME_END_GPU(device_time)
        size = N_EQUATIONS * sizeof(float);

    CUDA_CALL(hipGetLastError())

        CUDA_CALL(hipDeviceSynchronize())
        CUDA_CALL(hipMemcpy(X0_GPU, d_X0, size, hipMemcpyDeviceToHost))

        CUDA_CALL(hipMemcpy(X1_GPU, d_X1, size, hipMemcpyDeviceToHost))

        CUDA_CALL(hipMemcpy(FX0_GPU, d_FX0, size, hipMemcpyDeviceToHost))

        CUDA_CALL(hipMemcpy(FX1_GPU, d_FX1, size, hipMemcpyDeviceToHost))
       

        CHECK_TIME_DEST_GPU();

Error:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_X0);
    hipFree(d_X1);
    hipFree(d_FX0);
    hipFree(d_FX1);

    return cudaStatus;
}